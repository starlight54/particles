#include "hip/hip_runtime.h"
#include "mdIterator.h"
#include "mdIterator.cuh"

void MolDynIterator::UpdatePositions(ParticleSystem* particles)
{
	DeviceUpdatePositions << < particles.numBlocks,
		particles->numThreadsPerBlock >> > (particles, integrator,
		devicePos, deviceVel, deviceForce, deltaT, maxX, maxY, maxZ, kinEn);
}

void MolDynIterator::UpdateForces(unsigned long numParticles)
{
	DeviceUpdateForces << < particles.numBlocks,
		particles->numThreadsPerBlock >> > (particles->numParticles, deviceForce,
		devicePos, maxX, maxY, maxZ, forceEvaluator);
}

void MolDynIterator::UpdateVelocitiesT(unsigned long numParticles)
{
	UpdateVelocitiesT << < particles.numBlocks,
		particles->numThreadsPerBlock >> > (particles->numParticles, deviceForce,
		deltaT, deviceVel);
}

__global__ void DeviceUpdatePositions(ParticleSystem* particles, IIntegrationEvaluator* integrator, 
	double* devicePos, double* deviceVel, double* deviceForce, double deltaT, double maxX,
	double maxY, double maxZ, double &kinEn)
{
	int c = 0;

	for (int i = 0; i < 3 * particles->numParticles; i++) {

		if (c == 3) {
			c = 0;
		}

		double newPos = integrator->Evaluate(devicePos[i], deviceVel[i],
			deviceForce[i], deltaT);
		/*
		double tempXrX = particles->pos[i * 3 + 0] - prevPos[i * 3 + 0];
		double tempYrY = particles->pos[i * 3 + 1] - prevPos[i * 3 + 1];
		double tempZrZ = particles->pos[i * 3 + 2] - prevPos[i * 3 + 2];

		tempXrX = tempXrX - (maxX * (round(tempXrX / maxX)));
		tempYrY = tempYrY - (maxY * (round(tempYrY / maxY)));
		tempZrZ = tempZrZ - (maxZ * (round(tempZrZ / maxZ)));

		xrX = (2 * particles->pos[i * 3 + 0]) - (particles->
		pos[i * 3 + 0] - tempXrX) + (pow(deltaT, 2) *
		force[i * 3 + 0]);
		yrY = (2 * particles->pos[i * 3 + 1]) - (particles->
		pos[i * 3 + 1] - tempYrY) + (pow(deltaT, 2) *
		force[i * 3 + 1]);
		zrZ = (2 * particles->pos[i * 3 + 2]) - (particles->
		pos[i * 3 + 2] - tempZrZ) + (pow(deltaT, 2) *
		force[i * 3 + 2]);

		vel[i * 3 + 0] = (xrX - (particles->pos[i * 3 + 0] -
		tempXrX)) / (2 * deltaT);
		vel[i * 3 + 1] = (yrY - (particles->pos[i * 3 + 1] -
		tempYrY)) / (2 * deltaT);
		vel[i * 3 + 2] = (zrZ - (particles->pos[i * 3 + 2] -
		tempZrZ)) / (2 * deltaT);

		double tempXrX = xrX - prevPos[i * 3 + 0];
		double tempYrY = yrY - prevPos[i * 3 + 1];
		double tempZrZ = zrZ - prevPos[i * 3 + 2];

		vel[i * 3 + 0] = (tempXrX) / (2 * deltaT);
		vel[i * 3 + 1] = (tempYrY) / (2 * deltaT);
		vel[i * 3 + 2] = (tempZrZ) / (2 * deltaT);

		*/

		/*
		prevPos[i * 3 + 0] = particles->pos[i * 3 + 0];
		prevPos[i * 3 + 1] = particles->pos[i * 3 + 1];
		prevPos[i * 3 + 2] = particles->pos[i * 3 + 2];
		*/
		double boundaryWidth = 0;

		if (c == 0) {
			boundaryWidth = maxX;
		} else if (c == 1) {
			boundaryWidth = maxY;
		} else {
			boundaryWidth = maxZ;
			double velX = deviceVel[i - 2];
			double velY = deviceVel[i - 1];
			double velZ = deviceVel[i];
			kinEn += (velX * velX + velY * velY + velZ * velZ);
		}

		newPos = fmod(newPos, boundaryWidth);
		newPos = newPos < 0 ? boundaryWidth + newPos : newPos;

		devicePos[i] = newPos;
		++c;

		//comVel[c++] += vel[i];
	}
}

__global__ void DeviceUpdateForces(unsigned long numParticles, double* deviceForce,
	double* devicePos, double maxX, double maxY, double maxZ, IForceEvaluator* forceEvaluator)
{
	for (int i = 0; i < 3 * numParticles; i++) {
		deviceForce[i] = 0;
	}

	for (int i = 0; i < 3 * numParticles - 1; i += 3) {
		for (int j = i + 3; j < 3 * numParticles; j += 3) {
			double xDist = devicePos[j] - devicePos[i];
			double yDist = devicePos[j + 1] - devicePos[i + 1];
			double zDist = devicePos[j + 2] - devicePos[i + 2];

			xDist = xDist - (maxX * round(xDist / maxX));
			yDist = yDist - (maxY * round(yDist / maxY));
			zDist = zDist - (maxZ * round(zDist / maxZ));

			if (forceEvaluator->CheckCutoff(xDist, yDist, zDist)) {

				double scaledForce = forceEvaluator->
					EvaluateScaledForce();
				deviceForce[i] -= scaledForce * xDist;
				deviceForce[j] += scaledForce * xDist;
				deviceForce[i + 1] -= scaledForce * yDist;
				deviceForce[j + 1] += scaledForce * yDist;
				deviceForce[i + 2] -= scaledForce * zDist;
				deviceForce[j + 2] += scaledForce * zDist;

				forceEvaluator->EvaluateEnergy();
			}
		}
	}

	for (int i = 0; i < 3 * numParticles; i++) {
		deviceForce[i] = deviceForce[i] * 24;
	}
}

__global__ void DeviceUpdateVelocitiesT(unsigned long numParticles, double* deviceForce,
	double deltaT, double* deviceVel)
{
	for (int i = 0; i < 3 * numParticles; ++i) {
		double forceT = deviceForce[i] * deltaT * 0.5;
		deviceVel[i] += forceT;
	}
}